#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <glm/glm.hpp>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_new.h>
#include <thrust/device_free.h>

#include "Window.h"
#include "cuda_errors.h"
#include "FrameBuffer.h"
#include "Ray.h"
#include "Sphere.h"
#include "World.h"
#include "Camera.h"
#include "BVHNode.h"
#include "raytracer/kernel.h"

//–– GPU kernels ––

// Initialize per-pixel RNG states
__global__ void initRng(hiprandState* rngStates, int resX, int resY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= resX || y >= resY) return;
    int idx = y * resX + x;
    hiprand_init(1984, idx, 0, &rngStates[idx]);
}

// Build scene and camera once
__global__ void setupScene(World** worldPtr, Camera** camPtr, CameraInfo camInfo) {
    if (threadIdx.x != 0 || blockIdx.x != 0) return;
    World* w = new World();
    w->add(new Sphere({0, 0, -1}, 0.5f, new Lambertian({0.8f,0.3f,0.3f})));
    w->add(new Sphere({-1.01f, 0, -1}, 0.5f, new Dielectric(1.5f)));
    w->add(new Sphere({-1, 10, -1}, 0.5f, new Dielectric(1.5f)));
    w->add(new Sphere({1, 0, -1}, 0.5f, new Metal({0.8f,0.8f,0.8f}, 0.3f)));
    w->add(new Sphere({0, -1000.5f, 0}, 1000.0f,
        new Lambertian(new CheckerTexture({0.2f,0.3f,0.1f},{0.9f}))));
    *worldPtr = w;
    *camPtr   = camInfo.construct_camera();
}

// Trace rays and write pixels
__global__ void traceRays(FrameBuffer fb, World** worldPtr, Camera** camPtr, hiprandState* rngStates) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= fb.width || y >= fb.height) return;
    int idx = y * fb.width + x;

    hiprandState rng = rngStates[idx];
    glm::vec3 accumColor{0.0f};
    const int samples = 3;

    for (int s = 0; s < samples; ++s) {
        float u = (x + hiprand_uniform(&rng)) / float(fb.width);
        float v = (y + hiprand_uniform(&rng)) / float(fb.height);
        Ray ray = (*camPtr)->get_ray(u, v);
        accumColor += fb.color(ray, **worldPtr, &rng);
    }
    rngStates[idx] = rng;

    accumColor /= float(samples);
    accumColor = glm::sqrt(accumColor);  // gamma correction
    fb.writePixel(x, y, glm::vec4(accumColor, 1.0f));
}

// Clean up device objects
__global__ void cleanupScene(World** worldPtr, Camera** camPtr) {
    delete *worldPtr;
    delete *camPtr;
}

//–– Host-side RenderKernel class ––

KernelInfo::KernelInfo(hipGraphicsResource_t gfxRes, int width, int height)
    : resources(gfxRes), resX(width), resY(height)
{
    fb = new FrameBuffer(resX, resY);
    camInfo = CameraInfo({0,0,0},{0,0,0},90.0f,float(resX),float(resY));

    // allocate on device
    thrust::device_new<Camera*   >(&devCamera);
    thrust::device_new<hiprandState>(resX*resY, &rngStates);
    thrust::device_new<World*    >(&devWorld);

    // build scene & camera
    setupScene<<<1,1>>>(&devWorld, &devCamera, camInfo);
    check_cuda_errors(hipDeviceSynchronize());

    // init RNG
    dim3 block{8,8};
    dim3 grid{(resX+7)/8,(resY+7)/8};
    initRng<<<grid,block>>>(rngStates, resX, resY);
    check_cuda_errors(hipDeviceSynchronize());
}

void KernelInfo::resize(int width, int height) {
    resX = width; resY = height;
    delete fb;
    fb = new FrameBuffer(resX, resY);

    thrust::device_free(rngStates);
    thrust::device_new<hiprandState>(resX*resY, &rngStates);

    dim3 block{8,8}, grid{(resX+7)/8,(resY+7)/8};
    initRng<<<grid,block>>>(rngStates, resX, resY);
    check_cuda_errors(hipDeviceSynchronize());
}

void KernelInfo::set_camera(glm::vec3 pos, glm::vec3 dir, glm::vec3 up) {
    // single-thread update
    set_device_camera<<<1,1>>>(&devCamera, pos, dir, up, float(resX)/resY);
    check_cuda_errors(hipDeviceSynchronize());
}

void KernelInfo::render() {
    check_cuda_errors(hipGraphicsMapResources(1,&resources));
    check_cuda_errors(hipGraphicsResourceGetMappedPointer(
        (void**)&fb->device_ptr, &fb->buffer_size, resources));

    dim3 block{32,
